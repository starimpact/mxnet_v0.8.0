#include "hip/hip_runtime.h"
/*!
 * Copyright (c) 2016 by Contributors
 * \file postprocess_rpn.cu
 * \brief post process of rpn operator
 * \author Ming Zhang
*/
#include "./postprocess_rpn-inl.h"
#include "./mshadow_op.h"
#include <numeric>


namespace mshadow {

namespace cuda {


__global__ void PostProcessRPNForwardKernel(
                int count,
                const float *pfClsAll, const float *pfRegAll, 
                const float *pfAnchor, const float *pfOtherinfo, 
                int dwBatchNum, int dwAnchorNum, int dwFeatH, int dwFeatW, 
                float *pfBBsAll, float *pfScores, int dwMaxBBNum, int *pdwbb_num_all) {
#if 1     
  float clsthreshold = pfOtherinfo[0];
  int originalH = pfOtherinfo[1];
  int originalW = pfOtherinfo[2]; 
//  printf("clsthreshold:%.1f, originalH:%d, originalW:%d\n", clsthreshold, originalH, originalW);
//  __syncthreads();
  int index = (blockIdx.x + blockIdx.y * gridDim.x) * blockDim.x + threadIdx.x;

  if (1 && index < count)
  {
    int dwFeatSize = dwFeatH * dwFeatW;
    int dwFeatAnchorSize = dwFeatSize * dwAnchorNum;
    int dwBatchI = index / dwFeatAnchorSize;
    int dwAnchorI = (index - dwBatchI * dwFeatAnchorSize) / dwFeatSize;
    int dwRI = (index - dwBatchI * dwFeatAnchorSize - dwFeatSize * dwAnchorI) / dwFeatW;
    int dwCI = (index - dwBatchI * dwFeatAnchorSize - dwFeatSize * dwAnchorI) % dwFeatW;
    int dwOft = dwRI * dwFeatW + dwCI;
    int dwAnchorOft = dwAnchorI * dwFeatSize;
    const float *pfNowAnchor = pfAnchor + dwAnchorI * 2;
    const float *pfReg = pfRegAll + dwFeatAnchorSize * 4 * dwBatchI;
    const float *pfCls = pfClsAll + dwFeatAnchorSize * dwBatchI;
    float *pfBBs = pfBBsAll + dwBatchI * dwMaxBBNum * 5;
    float *pfScs = pfScores + dwBatchI * dwMaxBBNum;
    int *pdwbb_num_now = pdwbb_num_all + dwBatchI;

//    printf("bidxx:%d-bidxy:%d-gdimx:%d-bdimx:%d-tidxx:%d, index:%d, nownum:%d\n", blockIdx.x, blockIdx.y, gridDim.x, blockDim.x, threadIdx.x, index, nownum);
//    printf("bidxx:%d-bidxy:%d-gdimx:%d-bdimx:%d-tidxx:%d, index:%d\n", blockIdx.x, blockIdx.y, gridDim.x, blockDim.x, threadIdx.x, index);
//    __syncthreads();
#if 1
    float fScore = pfCls[dwOft + dwAnchorOft];
    if (fScore > clsthreshold)
    {
      int nownum = atomicInc((unsigned int*)(pdwbb_num_now), dwMaxBBNum);
      if (nownum < dwMaxBBNum)
      {
        float fCY = pfReg[dwAnchorOft * 4 + 0 * dwFeatSize + dwOft];
        float fCX = pfReg[dwAnchorOft * 4 + 1 * dwFeatSize + dwOft];
        float fH = pfReg[dwAnchorOft * 4 + 2 * dwFeatSize + dwOft];
        float fW = pfReg[dwAnchorOft * 4 + 3 * dwFeatSize + dwOft];
        fCY = fCY * pfNowAnchor[0] + ((float)(dwRI) * originalH) / dwFeatH;
        fCX = fCX * pfNowAnchor[1] + ((float)(dwCI) * originalW) / dwFeatW;
        fH = expf(fH) * pfNowAnchor[0];
        fW = expf(fW) * pfNowAnchor[1];
       
        {
          pfBBs[nownum * 5 + 0] = fScore;
          pfBBs[nownum * 5 + 1] = fCY;
          pfBBs[nownum * 5 + 2] = fCX;
          pfBBs[nownum * 5 + 3] = fH;
          pfBBs[nownum * 5 + 4] = fW;
          pfScs[nownum] = fScore;
//          printf("bidxx:%d-bidxy:%d-gdimx:%d-bdimx:%d-tidxx:%d, index:%d, nownum:%d\n", blockIdx.x, blockIdx.y, gridDim.x, blockDim.x, threadIdx.x, index, nownum);
//          __syncthreads();
        }
      }
    }
#endif
  }
#endif
}


inline void PostProcessRPNForward(const Tensor<gpu, 4> &datacls_in,
                           const Tensor<gpu, 4> &datareg_in,
                           const Tensor<gpu, 2> &anchorinfo_in,
                           const Tensor<gpu, 1> &otherinfo_in,
                           Tensor<gpu, 3> &bb_out) {
  CHECK_EQ(datacls_in.size(0), datareg_in.size(0));

  int dwBatchNum = datacls_in.size(0);
  int dwAnchorNum = anchorinfo_in.size(0);
  int bb_maxnum_per_batch = bb_out.size(1);
  
  int dwFeatH = datacls_in.size(2);
  int dwFeatW = datacls_in.size(3);

  int dwBufferPerLen = dwAnchorNum * dwFeatH * dwFeatW;

  Stream<gpu>* bbstream = bb_out.stream_;
  Tensor<gpu, 3, float> tBBBuffer(Shape3(dwBatchNum, dwBufferPerLen, 5));
  AllocSpace(&tBBBuffer, false);tBBBuffer.stream_ = bbstream;
  float *pfBBBuffer = tBBBuffer.dptr_;
  int dwBBBufferSize = dwBatchNum * dwBufferPerLen * 5;
  hipMemset(pfBBBuffer, 0, dwBBBufferSize*sizeof(float));

  Tensor<gpu, 2, float> tScores(Shape2(dwBatchNum, dwBufferPerLen));
  AllocSpace(&tScores, false);tScores.stream_ = bbstream;
  float *pfScores = tScores.dptr_;
  int dwScoreSize = dwBatchNum * dwBufferPerLen;
  hipMemset(pfScores, 0, dwScoreSize*sizeof(float));

  int dwBBMemLen = bb_out.MSize();
  hipMemset(bb_out.dptr_, 0, dwBBMemLen*sizeof(float));

  Tensor<gpu, 1, int> tCounter(Shape1(dwBatchNum));
  AllocSpace(&tCounter, false); tCounter.stream_ = bbstream;
  int *pdwCounter = tCounter.dptr_;
  hipMemset(pdwCounter, 0, dwBatchNum*sizeof(int));

  int count = dwFeatH * dwFeatW * dwAnchorNum * dwBatchNum;
  const int gridSize = (count + kMaxThreadsPerBlock - 1) / kMaxThreadsPerBlock;
  dim3 dimGrid(kMaxGridNum, (gridSize + kMaxGridNum - 1) / kMaxGridNum);
  dim3 dimBlock(kMaxThreadsPerBlock);

  CheckLaunchParam(dimGrid, dimBlock, "PostProcessRPN Forward");
  hipStream_t stream = Stream<gpu>::GetStream(bb_out.stream_);
  
  PostProcessRPNForwardKernel<<<dimGrid, dimBlock, 0, stream>>>(
            count, 
            datacls_in.dptr_, datareg_in.dptr_, 
            anchorinfo_in.dptr_, otherinfo_in.dptr_, dwBatchNum, dwAnchorNum, dwFeatH, dwFeatW, 
            pfBBBuffer, pfScores, dwBufferPerLen, pdwCounter);
//            bb_out.dptr_, pfScores, bb_maxnum_per_batch, pdwCounter);

#if 1
  {

    Tensor<cpu, 1, float> tRowScore_(Shape1(dwBufferPerLen));AllocSpace(&tRowScore_, false);
    Tensor<cpu, 1, int> tCounter_(Shape1(dwBatchNum));AllocSpace(&tCounter_, false);
    
    Copy<1, int>(tCounter_, tCounter, bbstream);
//    printf("mxnet=>dwCounter[%d, %d]:\n", dwBatchNum, dwBufferPerLen);
    for (int i = 0; i < dwBatchNum; i++)
    {
      Tensor<gpu, 2, float> tRowInfo = tBBBuffer[i];
      Tensor<gpu, 2, float> tRowOut = bb_out[i];
      Tensor<gpu, 1, float> tRowScore = tScores[i];
      Copy<1, float>(tRowScore_, tRowScore, bbstream);
      std::vector<int> index(dwBufferPerLen);
      std::iota(index.begin(), index.end(), 0);
      std::sort(index.begin(), index.end(),
                [&tRowScore_](size_t i0, size_t i1) {return tRowScore_[i0] > tRowScore_[i1];} );
//      printf("batch_%d:%d, \n", i, tCounter_[i]);
      int minnum = std::min(tCounter_[i], bb_maxnum_per_batch);
      for (int j = 0; j < minnum; j++)
      {
//        if (j < 20) printf("%f:%d, ", tRowScore_[index[j]], index[j]);
        Copy(tRowOut[j], tRowInfo[index[j]], bbstream);
      }
//      printf("\n");

   }
   FreeSpace(&tRowScore_);
   FreeSpace(&tCounter_);
  }
#endif
  FreeSpace(&tCounter);
  FreeSpace(&tBBBuffer);
  FreeSpace(&tScores);
  
}
  
} // namespace cuda

inline void PostProcessRPNForward(const Tensor<gpu, 4> &datacls_in,
                           const Tensor<gpu, 4> &datareg_in,
                           const Tensor<gpu, 2> &anchorinfo_in,
                           const Tensor<gpu, 1> &otherinfo_in,
                           Tensor<gpu, 3> &bb_out) {
//  printf("originalW:%d\n", originalW);                           
  cuda::PostProcessRPNForward(datacls_in, datareg_in, anchorinfo_in, otherinfo_in, bb_out);
}

} // namespace mshadow


namespace mxnet {
namespace op {

template<>
Operator *CreateOp<gpu>(PostProcessRPNParam param) {
  return new PostProcessRPNOp<gpu>(param);
}

}  // namespace op
}  // namespace mxnet
